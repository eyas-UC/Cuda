
#include <hip/hip_runtime.h>
#include <stdio.h>




/* A,B input arrays and C is the output and they all have the same size*/
__global__ void matrix_multiplication(int* A,int* B, int* C,
									  int Arow, int Acol,
									  int Brow, int Bcol)
{
	// get the flattened index
	int row = blockIdx.x * blockDim.x+ threadIdx.x;
	int col = blockIdx.y * blockDim.y+ threadIdx.y;
	// dimension mismatch!
	if (Acol != Brow)
		return;

	int val = 0;
	// boundry check for dimensions
 	if (row < Arow && col < Bcol)
	{
		for (int k = 0; k <Acol ; k++)
		{
			// this is the tricky part...
			// you probably need a pen a paper to get it right
			// fix row of the A and fix the col of B (Hint)
			// A[row*N + k]  --> fix row then travese its elements
			// B[col + k * N]--> fix column and traverse its elements
			val += A[row*Arow + k] * B[col + k * Bcol];
			C[row*Arow +col] += val;
			if (row==0 && col ==1)
			{
				printf("\n");
				printf("A[row*Arow + k]   = A[%i * %i + %i] = %i\n",row,Arow,k, A[row*Arow + k]);
				printf("B[col + k * Bcol] = B[%i + %i * %i] = %i\n",col,k,Bcol, B[col + k * Bcol]);
				printf("\n");
				// printf("C[row*Arow +col]  = C[%i * %i + %i] = %i\n",row,Arow,col, C[row*Arow +col]);
			}
		}
	}
}

void print_matrix(int * A, int row, int col)
{
	for(int i =0; i <(row*3+2);i++)
		printf("-");
	printf("\n");
	for(int i =0; i< row; i++)
	{
		printf("|");
		for(int j =0; j<col; j++)
		{
			printf(" %i ",A[i*row+j]);
		}
		printf("|\n");
	}
	for(int i =0; i <(row*3+2);i++)
		printf("-");
	printf("\n");

}
int main()
{
	// int N = (1<<2);
	int Arow = 4;
	int Acol = 3;
	int Brow = 3;
	int Bcol = 2;
	hipError_t syncE, asyncE;
	// create a pointer and allocate memory for it
	printf("starting things\n");
	int * arrA,*arrB, *arrC;
	// for a 2D array size will be N * N * size of int
	int sizeA = Arow * Acol * sizeof(int);
	int sizeB = Brow * Bcol * sizeof(int);
	int sizeC = Arow * Bcol * sizeof(int);
	hipMallocManaged(&arrA, sizeA);
	hipMallocManaged(&arrB, sizeB);
	hipMallocManaged(&arrC, sizeC);
	// initialization A
	for(int row = 0; row < Arow; row++)
	{
		for(int col = 0; col < Acol; col++)
		{
			arrA[row*Arow+col]=row;
		}
	}
	// initialization B
	for(int row = 0; row < Brow; row++)
	{
		for(int col = 0; col < Bcol; col++)
		{
			arrB[row*Bcol+col]=1;
		}
	}
	printf("sizeA=%i, sizeB=%i, sizeC=%i\n",sizeA,sizeB,sizeC);
	// define the dimention struct
	dim3 threads_per_block (8,8,1);
	dim3 no_block (8,8,1);
	//  run the cuda kernel;
	matrix_multiplication<<<no_block,threads_per_block>>>(arrA,arrB,arrC,Arow,Acol,Brow,Bcol);// setting lower threads and blocks than actual data
	syncE = hipGetLastError();
	asyncE = hipDeviceSynchronize();
	printf("%s",syncE != hipSuccess? "synchronous Error occured\n":"Great!...No synchronous Error\n");
	printf("%s",asyncE != hipSuccess? "asynchronous Error occured\n":"Great!...No asynchronous Error\n");

	print_matrix(arrA,Arow,Acol);
	print_matrix(arrB,Brow,Bcol);
	print_matrix(arrC,Arow,Bcol);
	// free allocated memory
	hipFree(arrA);
	hipFree(arrB);
	hipFree(arrC);
}
