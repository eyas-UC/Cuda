
#include <hip/hip_runtime.h>
#include <stdio.h>





__global__ void add_two(int* array,int array_size)
{
	// get the flattened index
	int i = blockIdx.x * blockDim.x+ threadIdx.x;
	int stride =gridDim.x * blockDim.x; //total no of threads in grid


	for (int k = i; k <array_size ; k+=stride)
	{
		array[k]+=2;
	}
}


int main()
{
	int N = 2<<10;
	// create a pointer and allocate memory for it
	printf("starting things\n");
	int * arr;
	size_t size = N * sizeof(int);
	hipMallocManaged(&arr, size);
	// set values with 2
	hipMemset(arr,0, size);
	printf("N=%i and size=%i\n",(int)N,(int)size);
	 // run the cuda kernel;
	add_two<<<8,16>>>(arr,N);// setting lower threads and blocks than actual data
	hipDeviceSynchronize();


	bool all_good = true;
	for(size_t i = 0; i<N;i++)
	{
		if(arr[i]!=2)
		{
			printf("arr[%i] = %i\n", (int)i,arr[i]);
			all_good =false;
			break;
		}
	}
	printf("%s",(all_good)?"all went well!\n":"something wrong\n" );
	hipFree(arr);
}
