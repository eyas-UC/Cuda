
#include <hip/hip_runtime.h>
#include <stdio.h>


__global__ void hello()
{
//# if __CUDA_ARCH__>=200
	printf("ThreadIDx=%i, BlockIdx=%i \n",threadIdx.x,blockIdx.x);

//#endif 
}

int main()
{
	
	hello<<<32,32>>>();
	hipDeviceSynchronize();
}
