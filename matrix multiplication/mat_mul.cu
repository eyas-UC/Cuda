
#include <hip/hip_runtime.h>
#include <stdio.h>




/* A,B input arrays and C is the output and they all have the same size*/
__global__ void matrix_multiplication(int* A,int* B, int* C, int N)
{
	// get the flattened index
	int val = 0;
	int row = blockIdx.x * blockDim.x+ threadIdx.x;
	int col = blockIdx.y * blockDim.y+ threadIdx.y;

	// boundry check for dimensions
 	if (row < N && col < N)
	{
		for (int k = 0; k <N ; k++)
		{
			// this is the tricky part...
			// you probably need a pen a paper to get it right
			// fix row of the A and fix the col of B (Hint)
			// A[row*N + k]  --> fix row then travese its elements
			// B[col + k * N]--> fix column and traverse its elements
			C[row*N +col] += A[row*N + k] * B[col + k * N];
		}
	}
}

void print_matrix(int * A, int row, int col)
{
	for(int i =0; i <(row*3+2);i++)
		printf("-");
	printf("\n");
	for(int i =0; i< row; i++)
	{
		printf("|");
		for(int j =0; j<col; j++)
		{
			printf(" %i ",A[i*row+j]);
		}
		printf("|\n");
	}
	for(int i =0; i <(row*3+2);i++)
		printf("-");
	printf("\n");

}
int main()
{
	int N = (1<<3) +1;
	hipError_t syncE, asyncE;
	// create a pointer and allocate memory for it
	printf("starting things\n");
	int * arrA,*arrB, *arrC;
	// for a 2D array size will be N * N * size of int
	size_t size = N * N * sizeof(int);
	hipMallocManaged(&arrA, size);
	hipMallocManaged(&arrB, size);
	hipMallocManaged(&arrC, size);
	// initialization
	for(int row = 0; row < N; row++)
	{
		for(int col = 0; col < N; col++)
		{
			arrA[row*N+col]=row;
			arrB[row*N+col]=row;
			arrC[row*N+col]= 0;
		}
	}
	printf("N=%i and size=%i\n",(int)N,(int)size);
	// define the dimention struct
	dim3 threads_per_block (8,8,1);
	dim3 no_block (8,8,1);
	//  run the cuda kernel;
	matrix_multiplication<<<no_block,threads_per_block>>>(arrA,arrB,arrC,N);// setting lower threads and blocks than actual data
	syncE = hipGetLastError();
	asyncE = hipDeviceSynchronize();
	printf("%s",syncE != hipSuccess? "synchronous Error occured\n":"Great!...No synchronous Error\n");
	printf("%s",asyncE != hipSuccess? "asynchronous Error occured\n":"Great!...No asynchronous Error\n");

	print_matrix(arrA,N,N);
	print_matrix(arrB,N,N);
	print_matrix(arrC,N,N);
	// free allocated memory
	hipFree(arrA);
	hipFree(arrB);
	hipFree(arrC);
}
