
#include <hip/hip_runtime.h>
#include <stdio.h>




/* A,B input arrays and C is the output and they all have the same size*/
__global__ void matrix_multiplication(int* A,int* B, int* C, int N)
{
	// get the flattened index
	int row = blockIdx.x * blockDim.x+ threadIdx.x;
	int col = blockIdx.y * blockDim.y+ threadIdx.y;

	for (int k = 0; k <N ; k++)
	{
		C[k] += A[row*N + k] + B[col *k +N];
	}
}

void print_matrix(int * A, int row, int col)
{
	for(int i =0; i< row; i++)
	{
		printf("|");
		for(int j =0; j<col; j++)
		{
			printf(" %i ",A[i*row+j]);
		}
		printf("|\n");

	}
}
int main()
{
	int N = 2<<2;
	hipError_t syncE, asyncE;
	// create a pointer and allocate memory for it
	printf("starting things\n");
	int * arrA,*arrB, *arrC;
	// for a 2D array size will be N * N * size of int
	size_t size = N * N * sizeof(int);
	hipMallocManaged(&arrA, size);
	hipMallocManaged(&arrB, size);
	hipMallocManaged(&arrC, size);
	// initialization
	for(int row = 0; row < N; row++)
	{
		for(int col = 0; col < N; col++)
		{
			arrA[row*N+col]=row;
			arrB[row*N+col]=col;
			arrC[row*N+col]= 0;
		}
	}
	printf("N=%i and size=%i\n",(int)N,(int)size);
	// define the dimention struct
	dim3 threads_per_block (16,16,1);
	dim3 no_block (16,16,1);
	 // run the cuda kernel;
	// matrix_multiplication<<<no_block,threads_per_block>>>(arrA,arrB,arrC,N);// setting lower threads and blocks than actual data
	// syncE = cudaGetLastError();
	// asyncE = cudaDeviceSynchronize();
	// printf("%s",syncE != cudaSuccess? "synchronous Error occured\n":"Great!...No synchronous Error\n");
	// printf("%s",asyncE != cudaSuccess? "asynchronous Error occured\n":"Great!...No asynchronous Error\n");

	print_matrix(arrA,N,N);
	// free allocated memory
	hipFree(arrA);
	hipFree(arrB);
	hipFree(arrC);
}
