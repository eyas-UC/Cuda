
#include <hip/hip_runtime.h>
#include <stdio.h>




/* A,B input arrays and C is the output and they all have the same size*/
__global__ void add_vectors(int* A,int* B, int* C, int array_size)
{
	// get the flattened index
	int i = blockIdx.x * blockDim.x+ threadIdx.x;
	int stride =gridDim.x * blockDim.x; //total no of threads in grid

	for (int k = i; k <array_size ; k+=stride)
	{
		C[k]=A[k] + B[k];
	}
}


int main()
{
	int N = 2<<8;
	hipError_t syncE, asyncE;
	// create a pointer and allocate memory for it
	printf("starting things\n");
	int * arrA,*arrB, *arrC;
	size_t size = N * sizeof(int);
	hipMallocManaged(&arrA, size);
	hipMallocManaged(&arrB, size);
	hipMallocManaged(&arrC, size);
	// initialization
	for(int j = 0; j < N; j++)
	{
		arrA[j]=10;
		arrB[j]=10;
		arrC[j]= 0;
	}
	printf("N=%i and size=%i\n",(int)N,(int)size);
	 // run the cuda kernel;
	add_vectors<<<8,16>>>(arrA,arrB,arrC,N);// setting lower threads and blocks than actual data
	syncE = hipGetLastError();
	asyncE = hipDeviceSynchronize();
	printf("%s",syncE != hipSuccess? "synchronous Error occured\n":"Great!...No synchronous Error\n");
	printf("%s",asyncE != hipSuccess? "asynchronous Error occured\n":"Great!...No asynchronous Error\n");

	bool all_good = true;
	for(size_t i = 0; i<N;i++)
	{
		if(arrC[i]!=20)
		{
			printf("arr[%i] = %i\n", (int)i,arrC[i]);
			all_good =false;
			break;
		}
	}
	printf("%s",(all_good)?"all went well!\n":"something wrong\n" );
	// free allocated memory
	hipFree(arrA);
	hipFree(arrB);
	hipFree(arrC);
}
