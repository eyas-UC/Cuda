
#include <hip/hip_runtime.h>
#include <stdio.h>





__global__ void add_two(int* array,int array_size)
{
	// get the flattened index
	int i = blockIdx.x * blockDim.x+ threadIdx.x;
	int stride =gridDim.x * blockDim.x; //total no of threads in grid


	for (int k = i; k <array_size ; k+=stride)
	{
		array[k]+=2;
	}
}


int main()
{
	int N = 2<<10;
	hipError_t syncE, asyncE;
	// create a pointer and allocate memory for it
	printf("starting things\n");
	int * arr;
	size_t size = N * sizeof(int);
	hipMallocManaged(&arr, size);
	// set values with 2
	hipMemset(arr,0, size);
	printf("N=%i and size=%i\n",(int)N,(int)size);
	 // run the cuda kernel;
	add_two<<<8,16>>>(arr,N);// setting lower threads and blocks than actual data
	syncE = hipGetLastError();
	asyncE = hipDeviceSynchronize();

	printf("%s",syncE != hipSuccess? "synchronous Error occured\n":"Great!...No synchronous Error\n");
	printf("%s",asyncE != hipSuccess? "asynchronous Error occured\n":"Great!...No asynchronous Error\n");

	bool all_good = true;
	for(size_t i = 0; i<N;i++)
	{
		if(arr[i]!=2)
		{
			printf("arr[%i] = %i\n", (int)i,arr[i]);
			all_good =false;
			break;
		}
	}
	printf("%s",(all_good)?"all went well!\n":"something wrong\n" );
	hipFree(arr);
}
